#include "hip/hip_runtime.h"
/*
 * @Description: 
 * @Date: 2020-11-25 13:28:14
 * @LastEditors: PengyuWang
 * @LastEditTime: 2020-12-07 16:32:47
 * @FilePath: /sampling/src/kernel.cu
 */
#include "gpu_graph.cuh"
#include "kernel.cuh"


__global__ void init_kernel_ptr(Sampler *sampler) {
  if (TID == 0) {
    sampler->result.setAddrOffset();
    for (size_t i = 0; i < sampler->result.hop_num; i++) {
      sampler->result.high_degrees[i].Init();
    }
  }
}


__device__ bool AddTillSize(uint *size,
                            size_t target_size) // T *array,       T t,
{
  uint old = atomicAdd(size, 1);
  if (old < target_size) {
    return true;
  }
  return false;
}